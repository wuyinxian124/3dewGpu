/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int N = 32;
// const int blocksize = 16;
//*/

__global__
void hello(int *a, int *b, int *c) {
	// threadinx.x 从0开始，最大值是blockDim.x 。
	//blockDim.x 限制是dimBlock（）的参数，第一个对应blockDim.x 第二个对blockDim.y 以此类推.
	//blockIdx.x 从0开始，最大值是dimGrid（）第一个参数限制 .y ,z 以此类推。
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.x * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.z + threadIdx.z;

	a[i] = i;
	b[j] = j;
	c[k] = k;

	/*	int m = blockIdx.z*blockDim.x + threadIdx.x;
	 int n = blockIdx.z*blockDim.y + threadIdx.y;*/
	/*	e[m] = m;
	 f[n] = n;*/

}
/*
__global__
void init(float *vpp, float *vss, float *density, int nx, int ny) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.x * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.z + threadIdx.z;
	vpp[i] = 2300.;
	vss[j] = 1232.;
	density[k] = 1.;
}*/
__global__
void testInit(float *vpp ){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	vpp[i] = 1.;

}
int main() {

	int b[N] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int a[N] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int c[N] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int *ad;
	int *bd;
	int *cd;
	/*	int b[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	 int a[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	 int *ad;
	 int *bd;
	 int c[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	 int d[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	 int *cd;
	 int *dd;*/
	int nx = 200, ny = 200, nz = 200;
	float *vpp, *density, *vss;
	float *vppd, *densityd, *vssd;

	const int isize = N * sizeof(int);
	const int nxyz = sizeof(float) * nz * ny * nx;

	vpp = (float*) malloc(sizeof(float) * nz * ny * nx);
//	density = (float*) malloc(sizeof(float) * nz * ny * nx);
//	vss = (float*) malloc(sizeof(float) * nz * ny * nx);
	hipMalloc((void**) &vppd, nxyz);
//	cudaMalloc((void**) &densityd, nxyz);
//	cudaMalloc((void**) &vssd, nxyz);
	hipMemcpy(vppd, vpp, nxyz, hipMemcpyHostToDevice);
//	cudaMemcpy(densityd, density, nxyz, cudaMemcpyHostToDevice);
//	cudaMemcpy(vssd, vss, nxyz, cudaMemcpyHostToDevice);

	hipMalloc((void**) &ad, isize);
	hipMalloc((void**) &bd, isize);
	hipMemcpy(ad, a, isize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);
	hipMalloc((void**) &cd, isize);
	hipMemcpy(cd, c, isize, hipMemcpyHostToDevice);
	/*	cudaMalloc( (void**)&ad, isize );
	 cudaMalloc( (void**)&bd, isize );
	 cudaMemcpy( ad, a, isize, cudaMemcpyHostToDevice );
	 cudaMemcpy( bd, b, isize, cudaMemcpyHostToDevice );
	 cudaMalloc( (void**)&cd, isize );
	 cudaMalloc( (void**)&dd, isize );
	 cudaMemcpy( cd, c, isize, cudaMemcpyHostToDevice );
	 cudaMemcpy( dd, d, isize, cudaMemcpyHostToDevice );*/

	/*	dim3 dimGrid(2, 2);
	 dim3 dimBlock(4, 2, 2);
	 hello<<<dimGrid, dimBlock>>>(ad,bd,cd,dd);*/
	dim3 dimGrid(2, 2);
	dim3 dimBlock(nx, ny, nz);
/*	init<<<3, dimBlock>>>(vppd, densityd, vssd, 200, 200);*/
	testInit<<<200,200>>>(vpp);
	dim3 dimGrid1(2, 2);
	dim3 dimBlock1(4, 4, 4);
	hello<<<2, dimBlock1>>>(ad, bd, cd);

//	cudaMemcpy(density, densityd, nxyz, cudaMemcpyDeviceToHost);
	hipMemcpy(vpp, vppd, nxyz, hipMemcpyDeviceToHost);
//	cudaMemcpy(vss, vssd, nxyz, cudaMemcpyDeviceToHost);
//	cudaFree(densityd);
	hipFree(vppd);
//	cudaFree(vssd);

	hipMemcpy(b, bd, isize, hipMemcpyDeviceToHost);
	hipMemcpy(a, ad, isize, hipMemcpyDeviceToHost);
	hipFree(bd);
	hipFree(ad);
	hipMemcpy(c, cd, isize, hipMemcpyDeviceToHost);
	hipFree(cd);
	/*	cudaMemcpy( b, bd, isize, cudaMemcpyDeviceToHost );
	 cudaMemcpy( a, ad, isize, cudaMemcpyDeviceToHost );
	 cudaFree( bd );
	 cudaFree( ad );
	 cudaMemcpy( c, cd, isize, cudaMemcpyDeviceToHost );
	 cudaMemcpy( d, dd, isize, cudaMemcpyDeviceToHost );
	 cudaFree( cd );
	 cudaFree( dd );
	 int i;*/
	/*
	 int e[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	 int f[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	 int *ed;
	 int *fd;
	 cudaMalloc( (void**)&ed, isize );
	 cudaMalloc( (void**)&fd, isize );
	 cudaMemcpy( ed, e, isize, cudaMemcpyHostToDevice );
	 cudaMemcpy( fd, f, isize, cudaMemcpyHostToDevice );
	 cudaFree( ed );
	 cudaFree( fd );
	 printf("\n");
	 for( i=0;i<N;i++){
	 printf("%d=%d ",i,e[i]);
	 }
	 printf("\n");
	 for( i=0;i<N;i++){
	 printf("%d=%d ",i,f[i]);
	 }
	 */

	/*
	 for( i=0;i<N;i++){
	 printf("%d=%d ",i,a[i]);
	 }
	 printf("\n");
	 for( i=0;i<N;i++){
	 printf("%d=%d ",i,b[i]);
	 }
	 printf("\n");
	 for( i=0;i<N;i++){
	 printf("%d=%d ",i,c[i]);
	 }
	 printf("\n");
	 for( i=0;i<N;i++){
	 printf("%d=%d ",i,d[i]);
	 }
	 */
	int i = 0;
/*	for (i = 0; i < N; i++) {
		printf("%d=%f ", i, density[i]);
	}*/
	printf("\n");
	for (i = 0; i < N; i++) {
		printf("%d=%f ", i, vpp[i]);
	}
/*	printf("\n");
	for (i = 0; i < N; i++) {
		printf("%d=%f ", i, vss[i]);
	}*/
	printf("\n");
	for (i = 0; i < N; i++) {
		printf("%d=%d ", i, a[i]);
	}
	printf("\n");
	for (i = 0; i < N; i++) {
		printf("%d=%d ", i, b[i]);
	}
	printf("\n");
	for (i = 0; i < N; i++) {
		printf("%d=%d ", i, c[i]);
	}
	return EXIT_SUCCESS;
}
